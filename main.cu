#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>
//#include <helper_functions.h>

// Image processing operations
#include "cannyCPU.h"

#define MAXPIXEL 255

void copyImageFromFile(FILE *srcImage, int *dstImage, int width, int height);
void dumpImageToFile(int *srcImage, char *dstName, int width, int height);
void addSquareToImage(int *srcImage, int width, int height, int position, int x, int y);

int main(int argc, char **argv){
  
  // File I/O set up
  FILE *inputFile, *templateFile;
  char inputFileName[20];
  
  // Performance control
  bool enabledGPU = true;
  bool enabledCPU = true;
  int threadsPerBlock = 256;
  
  // Image information
  int width, height, maxPValue;
  int tWidth, tHeight, *matchedPos;

  int i;

  // Host data items
  int *image, *tmplate;
  int *gradientMag,*cannyImage,thresh = 50;
  enum direction *gradientDir;

  // Device data items
  int *d_image, *d_gradientMag, *d_gradientDir;
  
  if (argc < 5){
    printf("Incorrect command line arguments\n");
    exit(0);
  }

  for(i=1; i < argc; i++)
  {
    if (strcmp(argv[i],"-f") == 0)
    {
      strcpy(inputFileName,argv[i+1]);
    }
    else if (strcmp(argv[i],"-gThresh") == 0)
    {
      thresh = atoi(argv[i+1]);
    }
    else if (strcmp(argv[i],"-t") == 0)
    {
      templateFile = fopen(argv[i+1],"r");
    }
  }

  inputFile = fopen(inputFileName,"r");

  fscanf(inputFile,"P2 #%*[^\n]\n%d %d %d", &width, &height, &maxPValue);
  if (maxPValue != MAXPIXEL) {
    printf("Incorrect max pixel value\n");
    exit(0);
  }

  fscanf(templateFile,"P2 #%*[^\n]\n%d %d %d", &tWidth, &tHeight, &maxPValue);
  if (maxPValue != MAXPIXEL) {
    printf("Incorrect max pixel value\n");
    exit(0);
  }

  // Allocate Items in Host Memory
  if ( (image = (int*)malloc(width*height*sizeof(int))) == 0 ){
    printf("Error allocating image\n");
    exit(0);
  }

  if ( (gradientMag = (int*)malloc(width*height*sizeof(int))) == 0 ){
    printf("Error allocating gradient magnitude array\n");
    exit(0);
  }

  if ( (gradientDir = (enum direction*)malloc(width*height*sizeof(enum direction))) == 0 ){
    printf("Error allocating gradient direction array\n");
    exit(0);
  }

  if ( (cannyImage = (int*)malloc(width*height*sizeof(int))) == 0 ){
    printf("Error allocating cannyImage array\n");
    exit(0);
  }

  if ( (tmplate = (int*)malloc(tWidth*tHeight*sizeof(int))) == 0 ){
    printf("Error allocating template array\n");
    exit(0);
  }

  // Allocate Items in Device Memory
  hipError_t cError;
  cError = hipMalloc((void**) &d_image, width*height*sizeof(int));
  if (cError != hipSuccess){
    printf("hipMalloc d_image returned error code %d, line(%d)\n",cError, __LINE__); 
    exit(0);
  }
  
  cError = hipMalloc((void**) &d_gradientMag, width*height*sizeof(int));
  if (cError != hipSuccess){
    printf("hipMalloc d_gradientMag returned error code %d, line(%d)\n",cError, __LINE__); 
    exit(0);
  }

  cError = hipMalloc((void**) &d_gradientDir, width*height*sizeof(int));
  if (cError != hipSuccess){
    printf("hipMalloc d_gradientDir returned error code %d, line(%d)\n",cError, __LINE__); 
    exit(0);
  } 

  copyImageFromFile(inputFile, image, width, height);
  copyImageFromFile(templateFile, tmplate, tWidth, tHeight);

  // Copy image to from host to device
  cError = hipMemcpy(d_image, image, width*height*sizeof(int), hipMemcpyDeviceToHost);
  if (cError != hipSuccess){
    printf("hipMemcpy (image -> d_image) returned error %d, line: %d\n", cError, __LINE__);
    exit(0);
  }

  // Potential TODO: Noise reduction ( Gaussian )
  
  // Find gradient magnitude and directions
  calcGradientCPU(image, gradientMag, gradientDir, width, height, thresh);
  if (enabledGPU) {    
    //calcGradientGPU(
  }
  dumpImageToFile(gradientMag, "out-gradient.pgm", width, height);

  // Thin edges using non-maximum suppression
  thinEdgesCPU(gradientMag, gradientDir, width, height);
  dumpImageToFile(gradientMag, "out-edgethin.pgm", width, height);

  // TODO: Double Threshold (BFS from definite edges over potential edges)
  connectivityCPU(gradientMag, cannyImage, width, height, 85, 125);
  dumpImageToFile(cannyImage, "out-connected.pgm", width, height);

  // TODO: Matching algorithms
  //        Template: Sum of absolute differences, (maybe) Geometric differences
  matchedPos = (int*)malloc(sizeof(int));
  templateMatchCPU(cannyImage, width, height, tmplate, tWidth, tHeight, matchedPos);
  if (matchedPos > 0) {
    addSquareToImage(cannyImage, width, height, *matchedPos, tWidth, tHeight);
  }
  dumpImageToFile(cannyImage, "out-template.pgm", width, height);


  free(image);
  free(gradientMag);
  free(gradientDir);
  free(cannyImage);
  free(tmplate);

  //Free Device Memory
  hipFree(d_image);
  hipFree(d_gradientMag);
  hipFree(d_gradientDir);

  fclose(inputFile);
  fclose(templateFile);
  return 0;
}

void copyImageFromFile(FILE *srcImage, int *dstImage, int width, int height){
  int i;
  for(i = 0; i < width*height; i++){
    fscanf(srcImage, "%d",&dstImage[i]);
  }
}

void dumpImageToFile(int *srcImage, char *dstName, int width, int height){
  FILE *dstImage = fopen(dstName, "w");
  int i;

  fprintf(dstImage,"P2\n%d %d\n%d\n", width, height, MAXPIXEL);
  for(i =0; i < width*height; i++){
    fprintf(dstImage,"%d\n",srcImage[i]);
  }
  fclose(dstImage);
}

void addSquareToImage(int *srcImage, int width, int height, int position, int x, int y){
  int i,j;
  if (position < 0 || position%width + x > width || position/width > height){
    //out of bounds
  }else{
    for (j=0; j < x; j++){
      if (position%width + j < width) srcImage[position + j] = 255;
      if (position/width + y < height) srcImage[position + y*width + j] = 255;
    }
    for (i=0; i < y; i++){
      if (position%width + x < width) srcImage[position + i*width + x] = 255;
      if (position/width + i < height) srcImage[position + i*width ] = 255;
    }
  }
}
